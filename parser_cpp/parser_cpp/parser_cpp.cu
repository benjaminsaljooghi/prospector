#include "hip/hip_runtime.h"
using namespace std;


// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
//#include <thrust/device_vector.h>
#include ""
//#include <hip/hip_runtime_api.h>

#include "hip/device_functions.h"

#include <iostream>
#include <stdio.h>

//#include <hip/hip_fp16.h>
//#include <fstream>
//#include <string>
//#include <map>

#include <vector>

//#include <algorithm>
//#include <optional>
//#include <functional>
//
#include "consts.h"
#include "Sequence.h"
#include <algorithm>
//#include "Crispr.h"
//#include <set>
//
//map<string, string> parse_fasta(string file_path)
//{
//    cout << "reading: " << file_path << endl;
//    ifstream input(file_path);
//    if (!input.good())
//    {
//        throw "Error opening " + file_path;
//    }
//
//    map<string, string> seqs;
//    string line, name, content;
//    while (getline(input, line))
//    {
//        if (line.empty() || line[0] == '>') // Identifier marker
//        {
//            if (!name.empty())
//            {
//                // Get what we read from the last entry
//                seqs[name] = content;
//                name.clear();
//            }
//            if (!line.empty())
//            {
//                name = line.substr(1);
//            }
//            content.clear();
//        }
//        else if (!name.empty())
//        {
//            if (line.find(' ') != string::npos) // Invalid sequence--no spaces allowed
//            {
//                name.clear();
//                content.clear();
//            }
//            else
//            {
//                content += line;
//            }
//        }
//    }
//    if (!name.empty())
//    {
//        // Get what we read from the last 
//        seqs[name] = content;
//    }
//
//    return seqs;
//}
//
//Sequence parse_single_seq(string file_path)
//{
//    map<string, string> seqs = parse_fasta(file_path);
//    string seq = seqs.begin()->second;
//    return Sequence(seq, 0);
//}
//
//vector<string> get_kmers(string sequence, int k)
//{
//    vector<string> seqs;
//    for (size_t i = 0; i < sequence.length() - k + 1; i++)
//    {
//        seqs.push_back(sequence.substr(i, k));
//    }
//    return seqs;
//}
//
//bool mutant(Sequence a, Sequence b)
//{
//    if (!ALLOW_DISCREPANT_LENGTHS && a.length() != b.length())
//    {
//        throw exception();
//    }
//
//    int len = min(a.length(), b.length());
//
//    int allowed_point_mutations = a.length() / 10;
//    int point_mutations = 0;
//
//    for (int i = 0; i < len; i++)
//    {
//        if (a[i] != b[i] && ++point_mutations > allowed_point_mutations)
//        {
//            return false;
//        }
//    }
//    return true;
//}
//
//optional<Crispr> discover_crispr(Sequence genome, Sequence dyad)
//{
//
//    Crispr crispr;
//    crispr.add_repeat(dyad);
//
//    int k = dyad.length();
//
//    // Upstream scan
//    int index = dyad.start() + k + SPACER_SKIP;
//    const int reset = SCAN_DOMAIN;
//    int countdown = reset;
//    while (countdown-- > 0)
//    {
//        if (index + k > genome.end())
//        {
//            break;
//        }
//        Sequence kmer = genome.subseq(index++, k);
//        if (mutant(dyad, kmer))
//        {
//            crispr.add_repeat(kmer);
//            index = kmer.start() + k + SPACER_SKIP;
//            countdown = reset;
//        }
//    }
//
//    // Downstream scan
//    index = dyad.start() - k - SPACER_SKIP;
//    countdown = reset;
//    while (countdown-- > 0)
//    {
//        if (index < genome.start())
//        {
//            break;
//        }
//        Sequence kmer = genome.subseq(index--, k);
//        if (mutant(dyad, kmer))
//        {
//            crispr.add_repeat(kmer);
//            index = kmer.start() - k - SPACER_SKIP;   
//            countdown = reset;
//        }
//
//    }
//
//    if (crispr.repeats.size() >= REPEATS_MIN)
//    {
//        crispr.sort_repeats();
//        return optional<Crispr>{crispr};
//    }
//
//    return nullopt;
//}
//
//set<Crispr> discover_crisprs(Sequence genome, vector<Sequence> dyads)
//{
//    set<Crispr> crisprs;
//
//    size_t num_bytes = sizeof(vector<Sequence>);
//    for (int i = 0; i < dyads.size(); i++)
//        num_bytes += sizeof(dyads[i]);
//    cout << "dyads total " << num_bytes << " bytes" << endl;
//
//    cout << "discovering CRISPRs from " << dyads.size() << " dyads." << endl;
//    for (int i = 0; i < dyads.size(); i++)
//    {
//        Sequence dyad = dyads[i];
//        cout << "\rexamining dyad " << i << "/" << dyads.size() - 1 << " with start " << dyad.start() << "/" << genome.length();
//        optional<Crispr> crispr = discover_crispr(genome, dyad);
//        if (crispr.has_value())
//        {
//            cout << " -> CRISPR discovered at consensus start " << dyad.start() << endl;
//            crisprs.insert(*crispr);
//        }
//    }
//    cout << endl;
//    return crisprs;
//}




// CUDA BEGIN


__device__ bool mutant(Sequence a, Sequence b)
{
    //if (!ALLOW_DISCREPANT_LENGTHS && a.length() != b.length())
    //{
    //    throw exception();
    //}

    int len = min(a.length(), b.length());

    int allowed_point_mutations = a.length() / 10;
    int point_mutations = 0;

    for (int i = 0; i < len; i++)
    {
        if (a[i] != b[i] && ++point_mutations > allowed_point_mutations)
        {
            return false;
        }
    }
    return true;
}


#define N 1000

__device__ Sequence crispr_dyads[N];
__device__ int crispr_dyad_count = 0;

__device__ int my_push_back(Sequence& seq)
{
    int insert_pt = atomicAdd(&crispr_dyad_count, 1);
    if (insert_pt >= N)
    {
        return -1;
    }
    crispr_dyads[insert_pt] = seq;
    return insert_pt;
}

__device__ void discover_crispr_cuda(Sequence genome, Sequence dyad)
{
    my_push_back(dyad);

    int k = dyad.length();

    // Upstream scan
    int index = dyad.start() + k + SPACER_SKIP;
    const int reset = SCAN_DOMAIN;
    int countdown = reset;
    while (countdown-- > 0)
    {
        if (index + k > genome.end())
        {
            break;
        }
        Sequence kmer = genome.subseq(index++, k);
        if (mutant(dyad, kmer))
        {
            //crispr.push_back(kmer);
            my_push_back(kmer);
            index = kmer.start() + k + SPACER_SKIP;
            countdown = reset;
        }
    }

    // Downstream scan
    index = dyad.start() - k - SPACER_SKIP;
    countdown = reset;
    while (countdown-- > 0)
    {
        if (index < genome.start())
        {
            break;
        }
        Sequence kmer = genome.subseq(index--, k);
        if (mutant(dyad, kmer))
        {
            my_push_back(kmer);
            index = kmer.start() - k - SPACER_SKIP;
            countdown = reset;
        }
    }

    //return crispr;
}

__global__ void kernel(Sequence genome)
{
    printf(genome[threadIdx.x])
}

int main()
{
    kernel KERNEL_ARGS2(2, 256) ();

    int dsize;
    hipMemcpyFromSymbol(&dsize, HIP_SYMBOL(dev_count), sizeof(int));
    if (dsize >= N)
    {
        printf("overflow error\n");
        return 1;
    }

    vector<Match> results(dsize);
    hipMemcpyFromSymbol(&(results[0]), dev_data, dsize * sizeof(Match));
    cout << "number of matches = " << dsize << endl;
    cout << "A  =  " << results[dsize - 1].A << endl;
    cout << "B  =  " << results[dsize - 1].B << endl;
    cout << "C  =  " << results[dsize - 1].C << endl;
}


    //string test_path = R"(P:\CRISPR\test_data\test.fasta)";
    //string aureus_path = R"(P:\CRISPR\bacteria\aureus.fasta)";
    //string pyogenes_path = R"(P:\CRISPR\bacteria\pyogenes.fasta)";

    //Sequence pyogenes = parse_single_seq(pyogenes_path);

    //vector<Sequence> dyads = pyogenes.dyads(30, 40);
    //set<Crispr> crisprs = discover_crisprs(pyogenes, dyads);
   
    //cout << "discovered CRISPRs: " << endl;
    //for (auto c : crisprs)
    //{
    //    cout << c.stringification() << endl;
    //}

    //return 0;

