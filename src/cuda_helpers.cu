#include "cuda_helpers.h"
#include <cstdio>

hipError_t checkCudaAlways(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    }
    return result;
}

hipError_t checkCuda(hipError_t result)
{
#if DEBUG == 1
    checkCudaAlways(result);
#endif
    return result;
}

void cudaWait()
{
    checkCuda ( hipDeviceSynchronize() );
}
