#include "hip/hip_runtime.h"
#include "prospector.h"


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_fp16.h"

#include <cassert>
#include <chrono>
#include <stdio.h>


#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define CUDA_CALLABLE_MEMBER
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

#define DEBUG 0

#define C_GRID 128
#define C_BLOCK 1024


hipError_t checkCuda(hipError_t result)
{
#if DEBUG == 1
    if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
    return result;
}

__device__ int __popc (unsigned int x);

__device__ unsigned char difference_gpu(const ui& _a, const ui& _b)
{
    ui _xor = (_a ^ _b);
    ui evenBits = _xor & 0xAAAAAAAAAAAAAAAAull;
    ui oddBits = _xor & 0x5555555555555555ull;
    ui comp = (evenBits >> 1) | oddBits;
    return __popc(comp);
}

__global__ void compute_qmap(const ui* genome_encoding, const ui genome_encoding_size, unsigned char* qmap)
{
    const ui thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const ui stride = blockDim.x * gridDim.x;

    for (ui query = thread_id; query < genome_encoding_size - 200; query += stride)
    {
        ui q = genome_encoding[query];
        for (ui i = 0; i < MAP_SIZE; i++)
        {
            ui t = genome_encoding[query + K_START + SPACER_SKIP + i];
            qmap[(query*MAP_SIZE) + i] = difference_gpu(q, t);
        }
    }
}


std::chrono::_V2::system_clock::time_point time()
{
    return std::chrono::high_resolution_clock::now();
}

std::chrono::_V2::system_clock::time_point time(std::chrono::_V2::system_clock::time_point start, const char* message)
{
    auto curr = time();
    printf("%ldms %s\n", std::chrono::duration_cast<std::chrono::milliseconds>(curr - start).count(), message);
    return curr;
}


__device__ ui scheme(const char c)
{
    switch (c)
    {
        case 'A':
            return 0;
        case 'C':
            return 1;
        case 'G':
            return 2;
        case 'T':
            return 3;
    }
}

__global__ void compute_encoding(const char* genome, ui* genome_encoding, ui genome_size, ui genome_encoding_size)
{
    const ui thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const ui stride = blockDim.x * gridDim.x;

    for (ui i = thread_id; i < genome_encoding_size; i += stride)
    {
        ui e = 0;
        for (int j = 0; j < SIZE; j++)
        {
            e |= scheme(genome[i + j]) << (j * BITS);
        }
        genome_encoding[i] = e;
    }
}



void Prospector::device_init()
{
    std::chrono::_V2::system_clock::time_point start = time();
    hipFree(0);
    time(start, "device init");
}


Prospector::Encoding Prospector::get_genome_encoding(const char* genome, ui genome_size)
{
    std::chrono::_V2::system_clock::time_point start = time();

    hipError_t er;
    
    char* d_genome;
    ui bytes_genome = sizeof(char) * genome_size;
    er = hipMalloc(&d_genome, bytes_genome); checkCuda(er);
    er = hipMemcpy(d_genome, genome, bytes_genome, hipMemcpyHostToDevice); checkCuda(er);

    ui* d_genome_encoding;
    ui genome_encoding_size = genome_size - SIZE + 1;
    ui bytes_genome_encoding = sizeof(ui) * genome_encoding_size; 
    er = hipMalloc(&d_genome_encoding, bytes_genome_encoding); checkCuda(er);

    compute_encoding KERNEL_ARGS3(C_GRID, C_BLOCK, 0) (d_genome, d_genome_encoding, genome_size, genome_encoding_size);

    ui* genome_encoding;
    er = hipHostMalloc(&genome_encoding, bytes_genome_encoding); checkCuda(er);

    hipDeviceSynchronize();

    er = hipMemcpy(genome_encoding, d_genome_encoding, bytes_genome_encoding, hipMemcpyDeviceToHost); checkCuda(er);

    time(start, "genome encoding total");


    Prospector::Encoding encoding;

    encoding.encoding = genome_encoding;
    encoding.d_encoding = d_genome_encoding;

    return encoding;
}


uc* Prospector::get_qmap(ui* d_encoding, ui genome_encoding_size)
{
    assert(K_START >= SIZE);
    hipError_t er; 
    std::chrono::_V2::system_clock::time_point start = time();


    ui bytes_qmap = sizeof(uc) * genome_encoding_size * MAP_SIZE;
    
    uc* d_qmap;
    
    er = hipMalloc(&d_qmap, bytes_qmap); checkCuda(er);
    start = time(start, "qmap malloc");

    uc* qmap;
    er = hipHostMalloc(&qmap, bytes_qmap); checkCuda(er);
    start = time(start, "qmap mallochost");

    compute_qmap KERNEL_ARGS3(C_GRID, C_BLOCK, 0) (d_encoding, genome_encoding_size, d_qmap);

    hipDeviceSynchronize();
    start = time(start, "kernel");

    er = hipMemcpy(qmap, d_qmap, bytes_qmap, hipMemcpyDeviceToHost); checkCuda(er);
    start = time(start, "qmap memcpy");

    hipFree(d_qmap);

    return qmap;
}


