#include "hip/hip_runtime.h"
#include "prospector.h"


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_fp16.h"


#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define CUDA_CALLABLE_MEMBER
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif












#define BITS 2
#define SIZE 16
#define MAP_SIZE 50

map<char, ui> scheme {
    {'A', 0},
    {'C', 1},
    {'G', 2},
    {'T', 3} 
};


ui encoded(const string& kmer)
{
    #if DEBUG == 1
    assert(kmer.size() == SIZE);
    #endif
    ui e = 0;
    for (int i = 0; i < kmer.size(); i++)
        e |= scheme.at(kmer[i]) << (i * BITS);
    return e;
}

ui* encoded_genome(const string& genome)
{
    double __start = omp_get_wtime();
    ui num = genome.size() - SIZE + 1;
    ui* encoding = (ui*) malloc(sizeof(ui) * num);
    #pragma omp parallel for
    for (ui i = 0; i < num; i++) encoding[i] = encoded(genome.substr(i, SIZE));
    done(__start, "genome encoding", "\t");
    return encoding;
}

__host__ ui difference_cpu(const ui& _a, const ui& _b)
{
    ui _xor = (_a ^ _b);
    ui evenBits = _xor & 0xAAAAAAAAAAAAAAAAull;
    ui oddBits = _xor & 0x5555555555555555ull;
    ui comp = (evenBits >> 1) | oddBits;
    return __builtin_popcount(comp);
}

__device__ unsigned char difference_gpu(const ui& _a, const ui& _b)
{
    ui _xor = (_a ^ _b);
    ui evenBits = _xor & 0xAAAAAAAAAAAAAAAAull;
    ui oddBits = _xor & 0x5555555555555555ull;
    ui comp = (evenBits >> 1) | oddBits;
    return __popc(comp);
}


void cwait()
{
    double start = omp_get_wtime();
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(err);
	}
    done(start, "kernel", "\t");
}


hipError_t checkCuda(hipError_t result)
{
#if DEBUG == 1
    if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
    return result;
}


#define C_GRID 128
#define C_BLOCK 1024

__global__ void compute_qmap(
    const ui* genome_encoding,
    const ui genome_encoding_size,
    unsigned char* qmap
)
{
    const ui thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const ui stride = blockDim.x * gridDim.x;

    // for each q, compute the mutation scores for the downrange MAP_SIZE indices, packed all into a vector
    // then iterate over those scores and form the crisprs

    // do the popcounts of the initiating 8-mers have <= (8/MUTANT_TOLERANCE_RATIO=1) popcount?
    // compute the popcounts of each query compared to the next MAP_SIZE indices
    // this should be an extremely efficient computation, and involves zero branching
    for (ui query = thread_id; query < genome_encoding_size - 200; query += stride)
    {
        ui q = genome_encoding[query];
        for (ui i = 0; i < MAP_SIZE; i++)
        {
            ui t = genome_encoding[query + K_START + SPACER_SKIP + i];
            qmap[(query*MAP_SIZE) + i] = difference_gpu(q, t);
        }
    }
}


vector<ui> q_substrate(unsigned char* qmap, ui genome_encoding_size)
{
    // how many qs in this map are containment oriented
    double start = omp_get_wtime();
    // ui count = 0;
    vector<ui> queries;
    for (ui query = 0; query < genome_encoding_size - 200; query++)
    {
        for (ui i = 0; i < MAP_SIZE; i++)
        {
            if (qmap[(query*MAP_SIZE) + i] <= (SIZE / MUTANT_TOLERANCE_RATIO)) // 1 because 8 / 5 = 1
            {
                queries.push_back(query);
                break;
            }
        }
    }
    done(start, "q_substrate", "\t");
    // printf("%d %zd\n", genome_encoding_size-200, queries.size());
    // return count;
    return queries;
}


bool mutant(const char* genome, const ui* genome_encoding, const ui& k, const ui& allowed_mutations, const ui& i, const ui& j)
{
    ui diff = 0;
    const ui chunks = k / SIZE;
    // may generate a lot of crisprs that are filtered later (expensive) given that SIZE is large (16) here.
    // option is to do a more accurate mutation calculation either using per-char post the chunk division
    // or to encode entire kmers up to MAP_SIZ into ull's to compute the difference efficiently.
    // post k=MAP_SIZ we can use the MAP_SIZ-long ull initially, and then compute a per-char difference afterwards.

    for (ui chunk = 0; chunk < chunks; chunk++)
    {
        ui _i = genome_encoding[i + (chunk * SIZE)];
        ui _j = genome_encoding[j + (chunk * SIZE)];
        diff += difference_cpu(_i, _j);
        if (diff > allowed_mutations)
        {
            return false;
        }
    }
    const ui checked_so_far = (chunks * SIZE);

    return diff <= checked_so_far / MUTANT_TOLERANCE_RATIO;




    // for (ui __i = checked_so_far; i < k; __i++)
    // {
        // diff += genome[i + checked_so_far + __i] == genome[j + checked_so_far + __i] ? 0 : 1; 
    // }
    // return diff <= allowed_mutations;
    
}


vector<vector<ui>> single_k_from_q_substrate(const char* genome, vector<ui> queries, ui* genome_encoding, const ui& k)
{
    vector<vector<ui>> crisprs;
    ui allowed_mutations = k / MUTANT_TOLERANCE_RATIO;

    for (ui _q = 0; _q < queries.size(); _q++)
    {
        ui q = queries[_q];

        vector<ui> crispr;
        crispr.push_back(q);

        ui bound = q + k + SPACER_SKIP;
        
        for (ui t = bound; t - bound <= SPACER_MAX; t++)
        {
            if (mutant(genome, genome_encoding, k, allowed_mutations, q, t))
            {
                crispr.push_back(t);
                bound = t + k + SPACER_SKIP;
                t = bound;
            }
        }
        crisprs.push_back(crispr);
    }
    return crisprs;
}


void debug_map()
{
    // ui query = 1283501;
    // ui q = genome_encoding[query];
    // for (ui i = 0; i < 1000; i++)
    // {
    //     ui pos = query + K_START + SPACER_SKIP + i;
    //     ui diff = difference_cpu(genome_encoding[query], genome_encoding[pos]);

    //     printf("%s %d %d\n", genome.substr(pos, SIZE).c_str(), pos, diff);
    // }
}



vector<Crispr> prospector_main_gpu(const string& genome)
{

    hipDeviceReset();

    hipError_t er;

    ui* genome_encoding = encoded_genome(genome);
    ui genome_encoding_size = genome.size() - SIZE + 1;


    // char* device_genome;
    ui* device_genome_encoding;

    double start = omp_get_wtime();
    er = hipMalloc(&device_genome_encoding, 4 * genome_encoding_size); checkCuda(er);
    er = hipMemcpy(device_genome_encoding, &genome_encoding[0], 4 * genome_encoding_size, hipMemcpyHostToDevice); checkCuda(er);
    ui count_qmap = genome_encoding_size * MAP_SIZE;
    ui bytes_qmap = 1 * count_qmap;
    unsigned char* qmap, *device_qmap;
    er = hipHostMalloc(&qmap, bytes_qmap); checkCuda(er);
    er = hipMalloc(&device_qmap, bytes_qmap); checkCuda(er);
    er = hipMemset(device_qmap, 0, bytes_qmap); checkCuda(er);
    done(start, "meminit", "\t");

    compute_qmap KERNEL_ARGS3(C_GRID, C_BLOCK, 0)
    (
        device_genome_encoding,
        genome_encoding_size,
        device_qmap
    );

    cwait();

    er = hipMemcpy(qmap, device_qmap, bytes_qmap, hipMemcpyDeviceToHost); checkCuda(er);

    hipFree(device_genome_encoding); hipFree(device_qmap);

    vector<ui> queries = q_substrate(qmap, genome_encoding_size);
    

    double total_single_k_time = 0;
    start = omp_get_wtime();
    vector<Crispr> all_crisprs;
    for (ui k = K_START; k < K_END; k++)
    {
        double __start = omp_get_wtime();
        vector<vector<ui>> crisprs = single_k_from_q_substrate(genome.c_str(), queries, genome_encoding, k);
        double __end = omp_get_wtime();
        total_single_k_time += __end - __start;

        for (vector<ui> c : crisprs)
        {
            if (c.size() >= MIN_REPEATS)
            {
                Crispr _c(k, c, c.size());
                all_crisprs.push_back(_c);   
            }
        }
    }
	printf("\t%.0fms total single k time\n", total_single_k_time * 1000.0);
    done(start, "crispr collection", "\t");

    return all_crisprs;
}


vector<Crispr> Prospector::prospector_main(const string& genome)
{
    assert(K_START >= SIZE);
    
    double start = omp_get_wtime(); 

    vector<Crispr> crisprs = prospector_main_gpu(genome);    
   
    done(start, "prospector");

    printf("%zd crisrps\n", crisprs.size());

    return crisprs;
}


