#include "hip/hip_runtime.h"
#include "prospector.h"


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_fp16.h"

#include <cassert>



#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define CUDA_CALLABLE_MEMBER
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

#define DEBUG 0

#define C_GRID 128
#define C_BLOCK 1024


hipError_t checkCuda(hipError_t result)
{
#if DEBUG == 1
    if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
    return result;
}


__device__ unsigned char difference_gpu(const ui& _a, const ui& _b)
{
    ui _xor = (_a ^ _b);
    ui evenBits = _xor & 0xAAAAAAAAAAAAAAAAull;
    ui oddBits = _xor & 0x5555555555555555ull;
    ui comp = (evenBits >> 1) | oddBits;
    return __popc(comp);
}

__global__ void compute_qmap(const ui* genome_encoding, const ui genome_encoding_size, unsigned char* qmap)
{
    const ui thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const ui stride = blockDim.x * gridDim.x;

    for (ui query = thread_id; query < genome_encoding_size - 200; query += stride)
    {
        ui q = genome_encoding[query];
        for (ui i = 0; i < MAP_SIZE; i++)
        {
            ui t = genome_encoding[query + K_START + SPACER_SKIP + i];
            qmap[(query*MAP_SIZE) + i] = difference_gpu(q, t);
        }
    }
}


unsigned char* Prospector::get_qmap(ui* genome_encoding, ui genome_encoding_size)
{
    assert(K_START >= SIZE);

    hipError_t er;

    ui* d_genome_encoding;

    er = hipMalloc(&d_genome_encoding, 4 * genome_encoding_size); checkCuda(er);
    er = hipMemcpy(d_genome_encoding, &genome_encoding[0], 4 * genome_encoding_size, hipMemcpyHostToDevice); checkCuda(er);
    ui count_qmap = genome_encoding_size * MAP_SIZE;
    ui bytes_qmap = 1 * count_qmap;
    unsigned char* qmap, *d_qmap;
    er = hipHostMalloc(&qmap, bytes_qmap); checkCuda(er);
    er = hipMalloc(&d_qmap, bytes_qmap); checkCuda(er);
    er = hipMemset(d_qmap, 0, bytes_qmap); checkCuda(er);

    compute_qmap KERNEL_ARGS3(C_GRID, C_BLOCK, 0) (d_genome_encoding, genome_encoding_size, d_qmap);

    hipDeviceSynchronize();

    er = hipMemcpy(qmap, d_qmap, bytes_qmap, hipMemcpyDeviceToHost); checkCuda(er);

    hipFree(d_genome_encoding); hipFree(d_qmap);

    return qmap;
}


