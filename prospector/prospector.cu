#include "hip/hip_runtime.h"
// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_fp16.h"


#include "../util/stdafx.h"
#include "../util/util.h"
#include "prospector.h"




#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define CUDA_CALLABLE_MEMBER
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif


void cwait()
{
	printf("waiting for kernel... ");
	clock_t start = clock();
	hipError_t err = hipDeviceSynchronize();
	printf("done in %.3f seconds\n", duration(start));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(err);
	}
}

void cufree(void* device_ptr)
{
	printf("executing cudafree... ");
    clock_t start = clock();
	hipError_t err = hipFree(device_ptr);
	if (err != hipSuccess)
	{
		fprintf(stderr, "failed to free device ptr (error code %s)!\n", hipGetErrorString(err));
		exit(err);
	}
    done(start);
}

template <typename T> void cpull(T* host, const T* device, unsigned int count)
{
	size_t bytes = count * sizeof(T);

	hipError_t err;

	printf("memcpy %*zd bytes from device... ", printf_BYTE_FORMAT_ALIGN, bytes);
    clock_t start = clock();
	err = hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "failed to copy from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(err);
	}
    done(start);
    
}

template <typename T> T* cpush(const T* src, unsigned int count)
{
	size_t bytes = count * sizeof(T);

	hipError_t err;
	T* ptr = NULL;

	printf("malloc+memcpy %*zd bytes to device... ", printf_BYTE_FORMAT_ALIGN, bytes);
    clock_t start = clock();
	err = hipMalloc((void**)& ptr, bytes);
	if (err != hipSuccess)
	{
		fprintf(stderr, "failed to malloc device (error code %s)!\n", hipGetErrorString(err));
		exit(err);
	}
    // done(start);

	// printf("memcpy %*zd bytes to device... ", printf_BYTE_FORMAT_ALIGN, bytes);
    // start = clock();
	err = hipMemcpy(ptr, src, bytes, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "failed to copy from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(err);
	}
    done(start);

	return (T*)ptr;
}

__device__ char complement(char nuc)
{
    // printf("%c\n", nuc);
    switch (nuc)
    {
    case 'A':
        return 'T';
    case 'T':
        return 'A';
    case 'C':
        return 'G';
    case 'G':
        return 'C';
    case 'N':
        return 'N';
    case 'n':
        return 'n';
    default:
        return 'n';
    }
}

__device__ bool mutant(const char* genome, unsigned int start_a, unsigned int start_b, unsigned int k)
{
	unsigned int allowed_mutations = k / MUTANT_TOLERANCE_RATIO;

	unsigned int mutations = 0;

	for (unsigned int i = 0; i < k; i++)
	{
        mutations += genome[start_a + i] == genome[start_b + i] ? 0 : 1;
		if (mutations > allowed_mutations)
        {
			return false;
        }
	}
	return true;
}

__device__ bool is_dyad_debug_check(unsigned int start_index)
{
    return start_index >= DEBUG_START && start_index <= DEBUG_END;
}

__device__ bool is_dyad(const char* genome, unsigned int start_index, unsigned int k)
{
    if (!is_dyad_debug_check(start_index))
    {
        return false;
    }

    unsigned int end_index = start_index + k - 1;

    unsigned int range = k/2 - 1;
    unsigned int mismatch_count = 0;
    for (unsigned int i = 0; i < range; i++)
	{
		char upstream = genome[start_index + i];
		char downstream = genome[end_index - i];
        mismatch_count += upstream == complement(downstream) ? 0 : 1;
	}

    double mismatch_ratio = (double) mismatch_count / (double) range;
    return mismatch_ratio < 0.75;
}


int atomicAdd(int* address, int val);
unsigned int atomicAdd(unsigned int* address,
                       unsigned int val);
unsigned long long int atomicAdd(unsigned long long int* address,
                                 unsigned long long int val);
float atomicAdd(float* address, float val);
double atomicAdd(double* address, double val);
__half2 atomicAdd(__half2 *address, __half2 val);
__half atomicAdd(__half *address, __half val);



__global__ void discover_crisprs(const char* genome, size_t genome_len, unsigned int* dyads, unsigned int dyad_count, unsigned int* buffer, unsigned int* buffer_index, unsigned int k)
{
    unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int query_d_index = thread_id; query_d_index < dyad_count; query_d_index += stride)
    {
        // printf("%d %d\n", thread_id, query_d_index);
        unsigned int query_dyad = dyads[query_d_index];
        unsigned int bound = query_dyad + k + SPACER_SKIP;

        unsigned int* local_crispr = new unsigned int[CRISPR_BUFFER];
        memset(local_crispr, 0, CRISPR_BUFFER * sizeof(unsigned int));

        local_crispr[0] = query_dyad;
        unsigned int repeat_index = 1;

        for (int target_d_index = query_d_index + 1; target_d_index < dyad_count; target_d_index++) // this for loop goes up to the dyad count but in practice this will never happen. May want to rewrite this. while loop? or for loop up to CRISPR_BUFFER?
        {
            unsigned int target_dyad = dyads[target_d_index];

            if (target_dyad < bound) continue;
            if (target_dyad - bound > SPACER_MAX) break;

            if (mutant(genome, query_dyad, target_dyad, k))
            {
                // printf("%d\n", repeat_index);
                local_crispr[repeat_index++] = target_dyad;
                bound = target_dyad + k + SPACER_SKIP;
            }
        }

        // repeat_index number represents count of elements in local_crifspr
        if (*(local_crispr + MIN_REPEATS) != 0)
        {
            // what we have are an array of dyads (local_crispr) which we need to "push_back" into a global crispr "vector".
            int buffer_start = atomicAdd(buffer_index, repeat_index + 1); // plus 1 because we need to leave a gap of 0 to separate the crisprs
            // printf("%d\n", buffer_start);
            memcpy(buffer + buffer_start, local_crispr, sizeof(unsigned int) * repeat_index);
            // *(buffer + repeat_index) = 0; this line isn't necessary because the initial memset for the global crispr sets the whole thing to 0
        }

        free(local_crispr);

    }
}





// https://devtalk.nvidia.com/default/topic/754830/atomic-counter-as-array-index-/

// __global__ void mykernel(int* nums, int* start)
// {
//     unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
//     unsigned int stride = blockDim.x * gridDim.x;
//     int i = atomicAdd (start, 1);
//     nums[i] = stride - i;
// }


// void foo()
// {
//     int grid = 2;
//     int block = 128;
//     int num = grid * block;
//     int* nums = new int[num];
//     memset(nums, 0, num * sizeof(int));
//     int* start = new int[1];
//     start[0] = 0;
//     int* device_nums = cpush(nums, num);
//     int* device_start = cpush(start, 1);
//     mykernel KERNEL_ARGS2(grid, block) (device_nums, device_start);
//     cwait();
//     cpull(nums, device_nums, num);
//     for (int i = 0; i < num; i++) printf("%d\n", nums[i]);
// }



vector<Crispr> crispr_gen(string genome, char* device_genome, size_t genome_len, vector<vector<unsigned int>> all_dyads)
{
    clock_t crispr_gen_start = clock();

    clock_t start;

    vector<Crispr> all_crisprs;

    for (size_t dyad_set = 0; dyad_set < all_dyads.size(); dyad_set++)
    {     
        unsigned int k = K_START + dyad_set;
        printf("for k %d\n", k);

        vector<unsigned int> dyads = all_dyads[dyad_set];
        // printf("dyad sort..."); start = clock();
        // sort(dyads.begin(), dyads.end());
        // done(start);
        unsigned int dyad_count = dyads.size(); printf("dyad count: %d\n", dyad_count);
        unsigned int* device_dyads = cpush(&dyads[0], dyad_count);

        printf("buffer..."); start = clock();
        // unsigned int crispr_buffer_count = dyad_count * CRISPR_BUFFER; printf("crispr buffer: %d\n", crispr_buffer_count);
        unsigned int crispr_buffer_count = 20000;
        unsigned int* crispr_buffer = new unsigned int[crispr_buffer_count];
        memset(crispr_buffer, 0, crispr_buffer_count * sizeof(unsigned int));
        done(start);

        unsigned int* device_crispr_buffer = cpush(crispr_buffer, crispr_buffer_count);


        unsigned int* crispr_buffer_index = new unsigned int[1];
        crispr_buffer_index[0] = 0;
        unsigned int* device_crispr_buffer_index = cpush(crispr_buffer_index, 1);

        discover_crisprs KERNEL_ARGS2(8, 256) 
                (device_genome, genome_len, device_dyads, dyad_count, device_crispr_buffer, device_crispr_buffer_index, k);    

        cwait();
        cpull(crispr_buffer, device_crispr_buffer, crispr_buffer_count);
        cpull(crispr_buffer_index, device_crispr_buffer_index, 1);
        cufree(device_crispr_buffer);
        cufree(device_crispr_buffer_index);

        printf("have crispr buffer index: %d\n", crispr_buffer_index[0]);

        if (crispr_buffer_index[0] >= crispr_buffer_count)
        {
            printf("BIG PROBLEM!!!!!!!!!!!!!\n");
        }

        vector<Crispr> k_crisprs;


        // printf("----begin crispr buffer-------\n");

        // for (unsigned int i = 0; i < crispr_buffer_index[0]; i++)
        // {
        //     printf("%d %d/%d %d\n", k, i, crispr_buffer_index[0], crispr_buffer[i]);
        //     std::this_thread::sleep_for(std::chrono::milliseconds(1));
        // }
        
        // printf("------end crispr buffer-----\n");

        printf("extract..."); start = clock(); 
        for (unsigned int i = 0; i < crispr_buffer_index[0]; i++)
        {
            unsigned int j;
            for (j = i; j < crispr_buffer_index[0]; j++)
            {
                if (*(crispr_buffer + j) == 0) break;
            }

            

            
            Crispr crispr(k, crispr_buffer + i, crispr_buffer + j);




            k_crisprs.push_back(crispr); 

            i = j + 1;
        }

        done(start);
        printf("insert..."); start = clock();
        all_crisprs.insert(all_crisprs.end(), k_crisprs.begin(), k_crisprs.end());
        done(start);

    }

    return all_crisprs;
}




__device__ void dyad_discovery_single_index(const char* genome, size_t genome_len, unsigned int d_index, unsigned int* dyad_buffer)
{
    for (unsigned int k = K_START; k < K_END; k++)
    {
        if (d_index + k < genome_len && is_dyad(genome, d_index, k))
        {
            unsigned int k_jump = genome_len;
            unsigned int k_index = k - K_START;
            dyad_buffer[k_index * k_jump + d_index] = d_index;
        }
    }
}


__global__ void dyad_discovery(const char* genome, size_t genome_len, unsigned int* dyad_buffer)
{
    unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int d_index = thread_id; d_index < genome_len; d_index += stride)
        dyad_discovery_single_index(genome, genome_len, d_index, dyad_buffer);
}



vector<unsigned int> dyad_lengths(vector<vector<unsigned int>> all_dyads)
{
	printf("compute dyad lengths... ");
	clock_t start = clock();
    vector<unsigned int> lengths;
	for (auto vec : all_dyads)
		lengths.push_back((unsigned int)vec.size());
    done(start);
	return lengths;
}

vector<vector<unsigned int>> dyad_gen(char* device_genome, size_t genome_len)
{
    size_t buffer_count = genome_len * (K_END - K_START);
    unsigned int* dyad_buffer = new unsigned int[buffer_count];
    memset(dyad_buffer, 0, buffer_count * sizeof(unsigned int));

    unsigned int* device_dyad_buffer = cpush(dyad_buffer, buffer_count);
    
    dyad_discovery KERNEL_ARGS2(16, 128) (device_genome, genome_len, device_dyad_buffer);
    cwait();

    cpull(dyad_buffer, device_dyad_buffer, buffer_count);
    cufree(device_dyad_buffer);

    printf("extract dyads... ");
    clock_t start = clock();
    vector<vector<unsigned int>> all_dyads;
    for (unsigned int k = K_START; k < K_END; k++)
    {
        unsigned int hopscotch = genome_len * (k - K_START);
        vector<unsigned int> dyads;
        for (unsigned int i = 0; i < genome_len; i++)
        {
            unsigned int hopscotch_leap = hopscotch + i;
            unsigned int dyad = dyad_buffer[hopscotch_leap];
            
            if (dyad != 0)
                dyads.push_back(dyad);
        }
        all_dyads.push_back(dyads);
    }
    done(start);

    return all_dyads;
}


void print_buffer(unsigned int total_dyad_count, unsigned int* crispr_buffer)
{    
    unsigned int count = 0;
    for (unsigned int d_index = 0; d_index < total_dyad_count; d_index++)
    {
        if (crispr_buffer[d_index * CRISPR_BUFFER + 1] == 0)
        {
            continue;
        }
        count += 1;

        printf("%d: ", d_index);
        for (unsigned int i = 0; i < CRISPR_BUFFER; i++)
        {
            printf("%d ", crispr_buffer[d_index * CRISPR_BUFFER + i]);
        }
        printf("\n");
    }
}

vector<Crispr> prospector_main_gpu(string genome)
{
    clock_t start;

    char* device_genome = cpush(genome.c_str(), genome.length());

    start = clock();
    vector<vector<unsigned int>> all_dyads = dyad_gen(device_genome, genome.length());
    done(start, "dyad_gen");
    
    start = clock();
    vector<Crispr> crisprs = crispr_gen(genome, device_genome, genome.length(), all_dyads);
    done(start, "crispr_gen");

    cufree(device_genome);

    return crisprs;
}


vector<Crispr> prospector_main(string genome)
{
    printf("genome has size %zd\n", genome.size());
    
    clock_t start;
    start = clock();
    vector<Crispr> crisprs = prospector_main_gpu(genome);
    done(start, "prospector");
    return crisprs;
}


